/******************************************************************************/
/* CUDA Sample Program (Vector add)                    monotone-RK 2014.08.21 */
/******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

void wait() {
  volatile int sum = 0;
  for (int i=0; i<10000; ++i) sum += i;
}

__global__
void vecadd(float *a, float *b, float *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(int argc, char *argv[]) {
  const int num = 16;

  float *a;
  float *b;
  float *c;

  hipHostMalloc(&a, num*sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&b, num*sizeof(float), hipHostMallocDefault);  
  hipHostMalloc(&c, num*sizeof(float), hipHostMallocDefault);

  for (int i=0; i<num; i++) {
    a[i] = 1.0;
    b[i] = 2.0;
    c[i] = 0.0;
  }
    
  dim3 grid_size  = dim3(1, 1, 1);    // determine the number of blocks
  dim3 block_size = dim3(num, 1, 1);  // determine the number of threads
  
  vecadd<<<grid_size, block_size>>>(a, b, c);
  
  wait();
  
  for (int i=0; i<num; ++i) printf("c[%2d]: %f\n", i, c[i]);
  
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);
  
  return 0;
}