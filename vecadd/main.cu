/******************************************************************************/
/* CUDA Sample Program (Vector add)                    monotone-RK 2014.08.21 */
/******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

__global__
void vecadd(float *a, float *b, float *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(int argc, char *argv[]) {
  const int num = 16;
  std::vector<float> a(num, 1.0);
  std::vector<float> b(num, 2.0);
  std::vector<float> c(num, 0.0);

  float *d_a;
  float *d_b;
  float *d_c;

  hipMalloc(&d_a, num * sizeof(float));
  hipMalloc(&d_b, num * sizeof(float));
  hipMalloc(&d_c, num * sizeof(float));

  hipMemcpy(d_a, &a[0], num*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b[0], num*sizeof(float), hipMemcpyHostToDevice);

  dim3 grid_size  = dim3(1, 1, 1);    // determine the number of blocks
  dim3 block_size = dim3(num, 1, 1);  // determine the number of threads
  
  vecadd<<<grid_size, block_size>>>(d_a, d_b, d_c);
  
  hipMemcpy(&c[0], d_c, num*sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  
  for (int i=0; i<num; i++) printf("c[%2d]: %f\n", i, c[i]);
  
  return 0;
}