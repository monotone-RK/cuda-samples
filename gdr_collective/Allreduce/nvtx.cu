// nvtx_wrappers

#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>

extern "C" {
void nvtxRangePushWrapper(const char *msg) {
    nvtxRangePushA(msg);
}

void nvtxRangePopWrapper() {
    nvtxRangePop();
}
}
